#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019-2021, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
 
 #include "LayerNormPlugin.h"
 #include <iostream>

using namespace nvinfer1;

PluginFieldCollection LayerNormPluginCreator::fc_{};
std::vector<PluginField> LayerNormPluginCreator::attr_;

template <int N> 
__global__ void layerNormKernel(float *pInput, float *pOutput)
{
    const int tx = threadIdx.x, index = blockIdx.x * N + threadIdx.x;

    __shared__ float temp[N/2];

    float value0 = pInput[index];
    float value1 = pInput[index + N/2];

    temp[tx] = value0 + value1;
    __syncthreads();

    for (int stride = N/4; stride >= 1; stride /= 2)
    {
        if (tx < stride)
        {
            temp[tx] += temp[tx + stride];
        }
        __syncthreads();
    }
    float mean = temp[0] / N;
    __syncthreads();

    temp[tx] = (value0 - mean) * (value0 - mean) + (value1 - mean) * (value1 - mean);
    __syncthreads();

    for (int stride = N/4; stride >= 1; stride /= 2)
    {
        if (tx < stride)
        {
            temp[tx] += temp[tx + stride];
        }
        __syncthreads();
    }
    float var = temp[0] / N;

    pOutput[index]       = (value0 - mean) * rsqrtf(var + 1e-5);
    pOutput[index + N/2] = (value1 - mean) * rsqrtf(var + 1e-5);
}

int32_t LayerNormPlugin::enqueue(const PluginTensorDesc* inputDesc, const PluginTensorDesc* outputDesc, const void* const* inputs, void* const* outputs, void* workspace, hipStream_t stream) noexcept
{
    const int nBlock = inputDesc[0].dims.d[0] * inputDesc[0].dims.d[1];
    const int nValuePerBlock = inputDesc[0].dims.d[inputDesc[0].dims.nbDims-1];

    std::cout << "nbDims: " << inputDesc[0].dims.nbDims << std::endl;
    std::cout << "nBlock: " << nBlock << std::endl;
    std::cout << "nValuePerBlock: " << nValuePerBlock << std::endl;
    // std::cout << "inputDesc[0].dims.d[2]: " << inputDesc[0].dims.d[2] << std::endl;
    // std::cout << "inputDesc[0].dims.d[3]: " << inputDesc[0].dims.d[3] << std::endl;

    switch(nValuePerBlock){
        case 128:
            layerNormKernel<128><<<nBlock, nValuePerBlock/2, 0, stream>>>((float *)inputs[0], (float *)outputs[0]);
            break;
        case 256:
            layerNormKernel<256><<<nBlock, nValuePerBlock/2, 0, stream>>>((float *)inputs[0], (float *)outputs[0]);
            break;
        case 512:
            layerNormKernel<512><<<nBlock, nValuePerBlock/2, 0, stream>>>((float *)inputs[0], (float *)outputs[0]);
            break;
        case 1024:
            layerNormKernel<1024><<<nBlock, nValuePerBlock/2, 0, stream>>>((float *)inputs[0], (float *)outputs[0]);
            break;
        default:
            std::cout << "LayerNorm plugin does not support nValuePerBlock = " << nValuePerBlock << std::endl;
            return -1;
    }

    std::cout << "end nBlock: " << nBlock << std::endl;
    std::cout << "end nValuePerBlock: " << nValuePerBlock << std::endl;
    
    return 0;
}

REGISTER_TENSORRT_PLUGIN(LayerNormPluginCreator);

